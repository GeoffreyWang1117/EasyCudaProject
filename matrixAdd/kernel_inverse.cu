﻿#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "matrix_utils.cuh"  // 包含打印矩阵的函数

static void invertMatrix(float* h_matrix, int n) {
    float* d_matrix, * d_invMatrix;
    int* P, * INFO;

    // 分配设备内存
    hipMalloc((void**)&d_matrix, n * n * sizeof(float));
    hipMalloc((void**)&d_invMatrix, n * n * sizeof(float));
    hipMalloc((void**)&P, n * sizeof(int));
    hipMalloc((void**)&INFO, sizeof(int));

    // 复制矩阵到设备
    hipMemcpy(d_matrix, h_matrix, n * n * sizeof(float), hipMemcpyHostToDevice);

    // 创建 cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // LU 分解
    hipblasSgetrfBatched(handle, n, &d_matrix, n, P, INFO, 1);

    // 求逆
    hipblasSgetriBatched(handle, n, (const float**)&d_matrix, n, P, &d_invMatrix, n, INFO, 1);

    // 复制结果回主机
    hipMemcpy(h_matrix, d_invMatrix, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // 释放资源
    hipFree(d_matrix);
    hipFree(d_invMatrix);
    hipFree(P);
    hipFree(INFO);
    hipblasDestroy(handle);
}

void testMatrixInverse(int size) {
    // 初始化方阵
    float* h_a = new float[size * size];

    std::cout << "Enter elements of the matrix (must be square):\n";
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            std::cin >> h_a[i * size + j];
        }
    }

    // 调用矩阵求逆
    invertMatrix(h_a, size);

    // 打印结果
    std::cout << "Inverse matrix:\n";
    printMatrix(h_a, size, size);

    // 释放内存
    delete[] h_a;
}
